#include "hip/hip_runtime.h"
//$nvcc -03 memTransfer.cu -o memTransfer
// nvprof ./memTransfer

#include <stdio.h>
#incldue <hip/hip_runtime.h>


__global__ void sumArrays(float *a, float *b, float *c, const int N)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;

	if(tid < N)
	{
		c[tid] = a[tid] + b[tid];
	}
}


int main(void)
{
	int dev = 0;
	hipSetDevice(0);

	unsigned int isize = 1<<22;
	unsigned int nbytes = sizeof(float) * isize;

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);

	printf("Starting: %s at", argv[0]);
	printf("device %d: memory size %d nbyte %5.2fMB\n", dev, deviceProp.name, isize, nbytes/(1024.0f * 1024.0f));

	
	//pageable host memory
	float *h_a = (float *)malloc(nbytes);

	float *d_a;
	hipMalloc((void **)&d_a, nbytes);

	for(unsgined int size = 0; i < isize; i++)
	{
		h_a[i] = 0.5f;
	} 

	hipMemcpy(d_a, h_a, nbytes, cudaMempcyHostToDevice);
	hipMemcpy(h_a, d_a, nbytes, hipMemcpyDeviceToHost);


	hipFree(d_a);
	free(h_a);

	hipDeviceReset();

	return EXIT_SUCCESS;
}		