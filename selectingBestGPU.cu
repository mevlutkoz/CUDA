#include <stdio.h>
#include <hip/hip_runtime.h>

int main(int argc, char **argv)
{
	int numDevices = 0;
	hipGetDeviceCount(&numDevices);
	
	if(numDevices > 1)
	{
		int maxMultiProcessors = 0, maxDevice = 0;
		for(int device = 0; device < numDevices; device++)
		{
			hipDeviceProp_t props,
			hipGetDeviceProperties(&props, device);

			if(maxMultiProcessors < props.multiProcessorCount)
			{
				maxMultiProcessors = props.multiProcessorCount;
				maxDevice = device;
			}

		}
		hipSetDevice(maxDevice);
	}


	return 0;
}
