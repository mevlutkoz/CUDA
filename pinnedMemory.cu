#include "hip/hip_runtime.h"
//$nvcc -03 pinnedMemory.cu -o pinnedMemory
// nvprof ./pinnedMemory

#include <stdio.h>
#incldue <hip/hip_runtime.h>


__global__ void sumArrays(float *a, float *b, float *c, const int N)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;

	if(tid < N)
	{
		c[tid] = a[tid] + b[tid];
	}
}


int main(void)
{
	int dev = 0;
	hipSetDevice(0);

	unsigned int isize = 1<<22;
	unsigned int nbytes = sizeof(float) * isize;

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);

	printf("Starting: %s at", argv[0]);
	printf("device %d: memory size %d nbyte %5.2fMB\n", dev, deviceProp.name, isize, nbytes/(1024.0f * 1024.0f));

	

	float *h_a_pinned;
	hipError_t status = hipHostMalloc((void **)&h_a_pinned, nbytes);
	if(status != hipSuccess)
	{
		fprintf(stderr, "Error allocating pinned host memory.\n");
		fprintf(stderr, Error code: %d error string:%s\n, status, hipGetErrorString(status));
		exit(1);
	} 

	float *d_a;
	hipMalloc((void **)&d_a, nbytes);

	for(unsgined int size = 0; i < isize; i++)
	{
		h_a[i] = 0.5f;
	} 

	hipMemcpy(d_a, h_a, nbytes, cudaMempcyHostToDevice);
	hipMemcpy(h_a, d_a, nbytes, hipMemcpyDeviceToHost);


	hipFree(d_a);
	hipHostFree(h_a_pinned);

	hipDeviceReset();

	return EXIT_SUCCESS;
}		