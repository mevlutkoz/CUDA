int deviceCount;

hipGetDeviceCount(&deviceCount);
int device;
for(device = 0; device < deviceCount; device++)
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device);
	printf("Device %d has compute capability %d.%d\n", deviceProp.major, deviceProp.minor);
}

